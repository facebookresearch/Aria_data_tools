#include "hip/hip_runtime.h"
/*
 * Copyright (c) Meta Platforms, Inc. and affiliates.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <Eigen/Dense>

#include <c10/cuda/CUDAGuard.h>

#include <eyetorch_kernels.h>
#include <models/dispatch.h>
#include <typedefs.h>

namespace eyetorch {

static constexpr int BLOCK_SIZE = 256;

inline int getGridSize(const int N) {
  return (N + BLOCK_SIZE - 1) / BLOCK_SIZE;
}

template <typename ScalarT, typename ProjectorT>
__global__ void project_forward_kernel(
    const typename TypeDefs<ScalarT, ProjectorT>::Point3D* points,
    const typename TypeDefs<ScalarT, ProjectorT>::CalibParamVec* calibParams,
    typename TypeDefs<ScalarT, ProjectorT>::Point2D* projections,
    const int N) {
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < N; i += blockDim.x * gridDim.x) {
    projections[i] = ProjectorT::project(points[i], calibParams[i]);
  }
}

template <typename ScalarT, typename ProjectorT>
void project_forward_gpu(
    const typename TypeDefs<ScalarT, ProjectorT>::Point3D* points,
    const typename TypeDefs<ScalarT, ProjectorT>::CalibParamVec* calibParams,
    typename TypeDefs<ScalarT, ProjectorT>::Point2D* projections,
    const int N) {
  project_forward_kernel<ScalarT, ProjectorT>
      <<<getGridSize(N), BLOCK_SIZE>>>(points, calibParams, projections, N);
}

template <typename ScalarT, typename ProjectorT>
__global__ void project_backward_kernel(
    const typename TypeDefs<ScalarT, ProjectorT>::Point3D* points,
    const typename TypeDefs<ScalarT, ProjectorT>::CalibParamVec* calibParams,
    const typename TypeDefs<ScalarT, ProjectorT>::Point2D* gradProjections,
    typename TypeDefs<ScalarT, ProjectorT>::Point3D* gradPoints,
    typename TypeDefs<ScalarT, ProjectorT>::CalibParamVec* gradParams,
    const int N) {
  using T = TypeDefs<ScalarT, ProjectorT>;

  using JPoint2DxPoint3D = typename T::JPoint2DxPoint3D;
  using JPoint2DxCalibParamVec = typename T::JPoint2DxCalibParamVec;

  JPoint2DxPoint3D jProjectionByPoint;
  JPoint2DxCalibParamVec jProjectionByParams;

  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < N; i += blockDim.x * gridDim.x) {
    ProjectorT::computeProjectJacobians(
        points[i], calibParams[i], jProjectionByPoint, jProjectionByParams);

    gradPoints[i] = jProjectionByPoint.transpose() * gradProjections[i];
    gradParams[i] = jProjectionByParams.transpose() * gradProjections[i];
  }
}

template <typename ScalarT, typename ProjectorT>
void project_backward_gpu(
    const typename TypeDefs<ScalarT, ProjectorT>::Point3D* points,
    const typename TypeDefs<ScalarT, ProjectorT>::CalibParamVec* calibParams,
    const typename TypeDefs<ScalarT, ProjectorT>::Point2D* gradProjections,
    typename TypeDefs<ScalarT, ProjectorT>::Point3D* gradPoints,
    typename TypeDefs<ScalarT, ProjectorT>::CalibParamVec* gradParams,
    const int N) {
  project_backward_kernel<ScalarT, ProjectorT><<<getGridSize(N), BLOCK_SIZE>>>(
      points, calibParams, gradProjections, gradPoints, gradParams, N);
}

template <typename ScalarT, typename ProjectorT>
__global__ void unproject_forward_kernel(
    const typename TypeDefs<ScalarT, ProjectorT>::Point2D* pixels,
    const typename TypeDefs<ScalarT, ProjectorT>::CalibParamVec* calibParams,
    typename TypeDefs<ScalarT, ProjectorT>::Point3D* rays,
    const int N) {
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < N; i += blockDim.x * gridDim.x) {
    rays[i] = ProjectorT::unproject(pixels[i], calibParams[i]);
  }
}

template <typename ScalarT, typename ProjectorT>
void unproject_forward_gpu(
    const typename TypeDefs<ScalarT, ProjectorT>::Point2D* pixels,
    const typename TypeDefs<ScalarT, ProjectorT>::CalibParamVec* calibParams,
    typename TypeDefs<ScalarT, ProjectorT>::Point3D* rays,
    const int N) {
  unproject_forward_kernel<ScalarT, ProjectorT>
      <<<getGridSize(N), BLOCK_SIZE>>>(pixels, calibParams, rays, N);
}

template <typename ScalarT, typename ProjectorT>
__global__ void unproject_backward_kernel(
    const typename TypeDefs<ScalarT, ProjectorT>::Point2D* pixels,
    const typename TypeDefs<ScalarT, ProjectorT>::CalibParamVec* calibParams,
    const typename TypeDefs<ScalarT, ProjectorT>::Point3D* gradRays,
    typename TypeDefs<ScalarT, ProjectorT>::Point2D* gradPixels,
    typename TypeDefs<ScalarT, ProjectorT>::CalibParamVec* gradParams,
    const int N) {
  using T = TypeDefs<ScalarT, ProjectorT>;

  using JPoint2DxPoint2D = typename T::JPoint2DxPoint2D;
  using JPoint2DxCalibParamVec = typename T::JPoint2DxCalibParamVec;

  // The projected ray is in 3D. Therefore, the Jacobians for the
  // pixel location and param vector should be 3x2 and 3xK,
  // respectively. However, because EyeTorch always unprojects
  // onto the unit plane, the partials w.r.t. the z-coordinate
  // are always zero. It is thus sufficient to compute the
  // 2x2 and 2xK top rows of the Jacobians, with the last row
  // being implicitly zero.
  JPoint2DxPoint2D jRaysByPixels;
  JPoint2DxCalibParamVec jRaysByParams;

  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < N; i += blockDim.x * gridDim.x) {
    ProjectorT::computeUnprojectJacobians(pixels[i], calibParams[i], jRaysByPixels, jRaysByParams);

    gradPixels[i] = jRaysByPixels.transpose() * gradRays[i].template head<2>();
    gradParams[i] = jRaysByParams.transpose() * gradRays[i].template head<2>();
  }
}

template <typename ScalarT, typename ProjectorT>
void unproject_backward_gpu(
    const typename TypeDefs<ScalarT, ProjectorT>::Point2D* pixels,
    const typename TypeDefs<ScalarT, ProjectorT>::CalibParamVec* calibParams,
    const typename TypeDefs<ScalarT, ProjectorT>::Point3D* gradRays,
    typename TypeDefs<ScalarT, ProjectorT>::Point2D* gradPixels,
    typename TypeDefs<ScalarT, ProjectorT>::CalibParamVec* gradParams,
    const int N) {
  unproject_backward_kernel<ScalarT, ProjectorT>
      <<<getGridSize(N), BLOCK_SIZE>>>(pixels, calibParams, gradRays, gradPixels, gradParams, N);
}

#define INSTANTIATE_GPU_FUNCTIONS(ProjectionModel)               \
  template void project_forward_gpu<float, ProjectionModel>(     \
      const typename TypeDefsF<ProjectionModel>::Point3D*,       \
      const typename TypeDefsF<ProjectionModel>::CalibParamVec*, \
      typename TypeDefsF<ProjectionModel>::Point2D*,             \
      const int);                                                \
  template void project_forward_gpu<double, ProjectionModel>(    \
      const typename TypeDefsD<ProjectionModel>::Point3D*,       \
      const typename TypeDefsD<ProjectionModel>::CalibParamVec*, \
      typename TypeDefsD<ProjectionModel>::Point2D*,             \
      const int);                                                \
  template void project_backward_gpu<float, ProjectionModel>(    \
      const typename TypeDefsF<ProjectionModel>::Point3D*,       \
      const typename TypeDefsF<ProjectionModel>::CalibParamVec*, \
      const typename TypeDefsF<ProjectionModel>::Point2D*,       \
      typename TypeDefsF<ProjectionModel>::Point3D*,             \
      typename TypeDefsF<ProjectionModel>::CalibParamVec*,       \
      const int);                                                \
  template void project_backward_gpu<double, ProjectionModel>(   \
      const typename TypeDefsD<ProjectionModel>::Point3D*,       \
      const typename TypeDefsD<ProjectionModel>::CalibParamVec*, \
      const typename TypeDefsD<ProjectionModel>::Point2D*,       \
      typename TypeDefsD<ProjectionModel>::Point3D*,             \
      typename TypeDefsD<ProjectionModel>::CalibParamVec*,       \
      const int);                                                \
  template void unproject_forward_gpu<float, ProjectionModel>(   \
      const typename TypeDefsF<ProjectionModel>::Point2D*,       \
      const typename TypeDefsF<ProjectionModel>::CalibParamVec*, \
      typename TypeDefsF<ProjectionModel>::Point3D*,             \
      const int);                                                \
  template void unproject_forward_gpu<double, ProjectionModel>(  \
      const typename TypeDefsD<ProjectionModel>::Point2D*,       \
      const typename TypeDefsD<ProjectionModel>::CalibParamVec*, \
      typename TypeDefsD<ProjectionModel>::Point3D*,             \
      const int);                                                \
  template void unproject_backward_gpu<float, ProjectionModel>(  \
      const typename TypeDefsF<ProjectionModel>::Point2D*,       \
      const typename TypeDefsF<ProjectionModel>::CalibParamVec*, \
      const typename TypeDefsF<ProjectionModel>::Point3D*,       \
      typename TypeDefsF<ProjectionModel>::Point2D*,             \
      typename TypeDefsF<ProjectionModel>::CalibParamVec*,       \
      const int);                                                \
  template void unproject_backward_gpu<double, ProjectionModel>( \
      const typename TypeDefsD<ProjectionModel>::Point2D*,       \
      const typename TypeDefsD<ProjectionModel>::CalibParamVec*, \
      const typename TypeDefsD<ProjectionModel>::Point3D*,       \
      typename TypeDefsD<ProjectionModel>::Point2D*,             \
      typename TypeDefsD<ProjectionModel>::CalibParamVec*,       \
      const int)

CALL_PREPROC_MACRO_FOR_EACH_PROJECTOR(INSTANTIATE_GPU_FUNCTIONS);

} // namespace eyetorch
